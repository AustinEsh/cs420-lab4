
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512
#define SIMPLE

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float input_s[BLOCK_SIZE];
    unsigned int segment = 2 * blockDim.x * blockIdx.x;
    unsigned int t = segment + threadIdx.x;
    unsigned int i = threadIdx.x;
    input_s[i] = in[t] + in[t + BLOCK_SIZE];

    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2){
        __syncthreads();
        if (i < stride){
            input_s[i] += input_s[i + stride];
        }
    }
    if (i == 0){
        atomicAdd(out, input_s[0]);
    }
}