
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE


/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
__global__ void preScan(float *out, float *in, unsigned n)
{
    extern __shared__ float temp[];  // allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2*thid] = in[2*thid]; // load input into shared memory
    temp[2*thid+1] = in[2*thid+1];
    for (int d = n>>1; d > 0; d >>= 1){                    // build sum in place up the tree
        __syncthreads();
        if (thid < d){ 
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;  
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    if (thid == 0) {
        temp[n - 1] = 0;
    } // clear the last element
    for (int d = 1; d < n; d *= 2){ // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d){ 
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1; 
 	        float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads(); 
    out[2*thid] = temp[2*thid]; // write results to device memory
    out[2*thid+1] = temp[2*thid+1]; 	
}